#include "hip/hip_runtime.h"
#include <stdlib.h>
#include <stdio.h>
#include <string.h>
#include <math.h>
#include <time.h>
#include <hip/hip_runtime.h>



#include "cuda_init.h"
#include "settingsJSON.cuh"
#include "cudaGenericAddressing.cuh"
#include "getPositionVector.cuh"

__global__ static void valSet(float * g_in)
{
	unsigned int lid = getGlobalIdx_1D_1D();
	g_in[lid] = (float)lid;
}

int main( int argc, char** argv) 
{

	gpuDeviceInit(findCudaDevice(argc,(const char**)argv));
	system("pause");
	
	cJSON * json;
	json = readJFile("settings.json");

	int nodes = cJSON_GetObjectItem(json,"count")->valueint;
	int incr = cJSON_GetObjectItem(json,"inc")->valueint;
	float * gpu;
	hipMalloc((void **)&gpu,nodes * sizeof(float));
	float * cpu = new float[nodes];
	valSet<<<nodes/128+1, 128>>>(gpu);
	hipMemcpy(cpu,gpu,nodes*sizeof(float),hipMemcpyDeviceToHost);
	for (int i = 0; i < nodes; i+= incr){
		printf("Val %f\n",cpu[i]);
	}
	system("pause");
	return 0;

}